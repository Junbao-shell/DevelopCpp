#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////
/// @copyright copyright description
/// 
/// @brief covolution module main in CUDA cufft
/// 
/// @file conv.cu
/// 
/// @author GaoJunbao(junbaogao@foxmail.com)
/// 
/// @date 2022-06-18
///////////////////////////////////////////////////////////

// Current Cpp header
#include "conv.h"
#include "conv.cuh"
// System header
// C/C++ standard library header
#include <iostream>
#include <string.h>
// External library header
// Current module header
#include "istring.h"
// Root directory header
#include "macro/cuda_check.h"

using namespace nmath;

Conv::Conv()
{
}

Conv::~Conv()
{
}

void Conv::Conv2D(const float *signal, 
                  const float *kernel, 
                  const int signal_dimx, 
                  const int signal_dimy, 
                  const int kernel_dimx, 
                  const int kernel_dimy,
                  float *result,
                  CONV_TYPE type)
{
    const int pad_dimx = signal_dimx + kernel_dimx - 1;
    const int pad_dimy = signal_dimy + kernel_dimy - 1;
    const int new_size = pad_dimx * pad_dimy;

    float *pad_signal, *pad_kernel, *pad_result;
    CUDACHECK(hipMalloc((void**)&pad_signal, sizeof(float) * new_size));
    CUDACHECK(hipMemset(pad_signal, 0, sizeof(float) * new_size));
    CUDACHECK(hipMalloc((void**)&pad_kernel, sizeof(float) * new_size));
    CUDACHECK(hipMemset(pad_kernel, 0, sizeof(float) * new_size));
    CUDACHECK(hipMalloc((void**)&pad_result, sizeof(float) * new_size));
    CUDACHECK(hipMemset(pad_result, 0, sizeof(float) * new_size));

    PadData2D(signal, signal_dimx, signal_dimy, pad_dimx, pad_dimy, pad_signal);
    PadData2D(kernel, kernel_dimx, kernel_dimy, pad_dimx, pad_dimy, pad_kernel);

    CUDACHECK(hipDeviceSynchronize());
    // create complex data
    hipfftComplex *cina, *cinb, *cout;
    CUDACHECK(hipMalloc((void**)&cina, sizeof(float) * new_size));
    CUDACHECK(hipMemset(cina, 0, sizeof(float) * new_size));
    CUDACHECK(hipMalloc((void**)&cinb, sizeof(float) * new_size));
    CUDACHECK(hipMemset(cinb, 0, sizeof(float) * new_size));
    CUDACHECK(hipMalloc((void**)&cout, sizeof(float) * new_size));
    CUDACHECK(hipMemset(cout, 0, sizeof(float) * new_size));

    // fft forward transform
    ForwardFFT2D(pad_signal, pad_dimx, pad_dimy, cina);
    ForwardFFT2D(pad_kernel, pad_dimx, pad_dimy, cinb);

    DotFFT2D(cina, cinb, pad_dimx, pad_dimy, cout);

    InverseFFT2D(cout, pad_dimx, pad_dimy, pad_result);

    if (CONV_TYPE::SAME == type)
    {
        GetSameData2D(pad_result, pad_dimx, pad_dimy, signal_dimx, signal_dimy, result);
    }
    else if (CONV_TYPE::VALID == type)
    {
        // TODO
        // GetValidData2D(pad_result, pad_dimx, pad_dimy, signal_dimx, signal_dimy, result);
        std::cout << "get valid zoo not implement" << std::endl;
    }
    else
    {
        CUDACHECK(hipMemcpy(result, pad_result, sizeof(float) * new_size, hipMemcpyDeviceToDevice));
    }

    CUDACHECK(hipFree(pad_signal)); pad_signal = nullptr;
    CUDACHECK(hipFree(pad_kernel)); pad_kernel = nullptr;
    CUDACHECK(hipFree(pad_result)); pad_result = nullptr;
    CUDACHECK(hipFree(cina)); cina = nullptr;
    CUDACHECK(hipFree(cinb)); cinb = nullptr;
    CUDACHECK(hipFree(cout)); cout = nullptr;
}

void Conv::DotFFT2D(const hipfftComplex *ina, 
                    const hipfftComplex *inb, 
                    const int dimx, 
                    const int dimy, 
                    hipfftComplex *out)
{
    const int size = dimx * dimy;

    dim3 Block(128);
    dim3 Grid((size + Block.x - 1) / Block.x);

    cudaComplexMatrixMul<<<Grid, Block>>>(ina, inb, size, out);
    CUDACHECK(hipGetLastError());
    CUDACHECK(hipDeviceSynchronize());
}

void Conv::ForwardFFT2D(float *in, const int dimx, const int dimy, hipfftComplex *out)
{
    hipfftHandle plan;
    hipfftPlan2d(&plan, dimy, dimx, HIPFFT_R2C);
    hipfftExecR2C(plan, in, out);
    hipfftDestroy(plan);
}

void Conv::InverseFFT2D(hipfftComplex *in, const int dimx, const int dimy, float *out)
{
    hipfftHandle plan;
    hipfftPlan2d(&plan, dimy, dimx, HIPFFT_C2R);
    hipfftExecC2R(plan, in, out);
    hipfftDestroy(plan);
}

void Conv::PadData2D(const float *data, 
                     const int dimx, 
                     const int dimy, 
                     const int pad_dimx, 
                     const int pad_dimy, 
                     float *pad_data)
{
    for (int i = 0; i < dimy; ++i)
    {
        const int raw_offset = i * dimx;
        const int pad_offset = i * pad_dimx;
        CUDACHECK(hipMemcpy(&pad_data[pad_offset], &data[raw_offset], sizeof(float) * dimx, hipMemcpyDeviceToDevice));
    }
}

void Conv::GetSameData2D(const float *pad_data, 
                         const int pad_dimx, 
                         const int pad_dimy, 
                         const int dimx, 
                         const int dimy, 
                         float *same_data)
{
    const int offsetx = std::ceil((pad_dimx - dimx) / 2.0);
    const int offsety = std::ceil((pad_dimy - dimy) / 2.0);

    for (int i = 0; i < dimy; ++i)
    {
        const int offset_full = (i + offsety) * pad_dimx + offsetx;
        const int offset_same = i * dimx;
        CUDACHECK(hipMemcpy(&same_data[offset_same], &pad_data[offset_full], sizeof(float) * dimx, 
            hipMemcpyDeviceToDevice));
    }
}
