#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////
/// @copyright copyright description
/// 
/// @brief cuda fft demo
/// 
/// @file cu_fft_demo.cu
/// 
/// @author GaoJunbao(junbaogao@foxmail.com)
/// 
/// @date 2022-06-14
///////////////////////////////////////////////////////////

// Current Cpp header
// System header
// C/C++ standard library header
#include <iostream>
#include <sstream>
#include <utility>
#include <chrono>
// External library header
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
// Current module header
// Root directory header

struct ComplexMultiply
{
    ComplexMultiply(int n) : N(n) {}

    __host__ __device__ hipfftComplex operator() (const hipfftComplex &a, const hipfftComplex &b)
    {
        hipfftComplex c;
        c.x = (a.x * b.x - a.y * b.y) / N;
        c.y = (a.x * b.y + a.y * b.x) / N;
        return c;
    }

    int N;
};

static __global__ void ComplexMulti(const hipfftComplex *a, const hipfftComplex *b, hipfftComplex *c, const int size)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        c[tid].x = (a[tid].x * b[tid].x - a[tid].y * b[tid].y) / size;
        c[tid].y = (a[tid].x * b[tid].y + a[tid].y * b[tid].x) / size;
    }
}

static __global__ void ComplexMulti2D(const hipfftComplex *a, const hipfftComplex *b, hipfftComplex *c, const int dimx, const int dimy)
{
    const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if (tidx < dimx && tidy < dimy)
    {
        const int i = tidy * dimx + tidx;

        c[i].x = (a[i].x * b[i].x - a[i].y * b[i].y) / (dimx * dimy);
        c[i].y = (a[i].x * b[i].y + a[i].y * b[i].x) / (dimx * dimy);
    }
}

template<typename T>
static inline std::string PrintArray(T *arr, const int size)
{
    std::stringstream stream;
    for (int i = 0; i < (size - 1); ++i)
    {
        stream << arr[i];
        stream << ' ';
    }
    stream << arr[size - 1];
    return stream.str();
}

template<typename T>
static inline std::string PrintGpuArray(T *arr, const int size)
{
    T *h_arr;
    hipHostMalloc((void**)&h_arr, sizeof(T) * size);
    hipMemcpy(h_arr, arr, sizeof(T) * size, hipMemcpyDeviceToHost);

    std::stringstream stream;
    for (int i = 0; i < (size - 1); ++i)
    {
        stream << h_arr[i];
        stream << ' ';
    }
    stream << h_arr[size - 1];

    hipHostFree(h_arr);
    h_arr = nullptr;

    return stream.str();
}

void ForwardFFT1D(float *in, hipfftComplex *out, const int size)
{
    hipfftHandle plan;
    hipfftPlan1d(&plan, size, HIPFFT_R2C, 1);
    hipfftExecR2C(plan, in, out);
    hipfftDestroy(plan);
}

void InverseFFT1D(hipfftComplex *in, float *out, const int size)
{
    hipfftHandle plan;
    hipfftPlan1d(&plan, size, HIPFFT_C2R, 1);
    hipfftExecC2R(plan, in, out);
    hipfftDestroy(plan);
}

void ForwardFFT2D(float *in, hipfftComplex *out, const int dimx, const int dimy)
{
    hipfftHandle plan;
    hipfftPlan2d(&plan, dimy, dimx, HIPFFT_R2C);
    hipfftExecR2C(plan, in, out);
    hipfftDestroy(plan);
}

void InverseFFT2D(hipfftComplex *in, float *out, const int dimx, const int dimy)
{
    hipfftHandle plan;
    hipfftPlan2d(&plan, dimy, dimx, HIPFFT_C2R);
    hipfftExecC2R(plan, in, out);
    hipfftDestroy(plan);
}

void Conv2DFFT(float *ina, float *inb, float *out, const int dimx, const int dimy)
{
    const int size = dimx * dimy;
    hipfftComplex *c_ina, *c_inb, *c_out;
    hipMalloc((void**)&c_ina, sizeof(hipfftComplex) * size);
    hipMalloc((void**)&c_inb, sizeof(hipfftComplex) * size);
    hipMalloc((void**)&c_out, sizeof(hipfftComplex) * size);

    std::cout << PrintGpuArray<float>(ina, size) << std::endl;

    ForwardFFT2D(ina, c_ina, dimx, dimy);
    ForwardFFT2D(inb, c_inb, dimx, dimy);
    hipDeviceSynchronize();

    hipMemset(ina, 0, sizeof(float) * size);
    InverseFFT2D(c_ina, ina, dimx, dimy);

    std::cout << PrintGpuArray<float>(ina, size) << std::endl;

    hipfftComplex *h_ina, *h_inb, *h_out;
    hipHostMalloc((void**)&h_ina, sizeof(hipfftComplex) * size);
    hipHostMalloc((void**)&h_inb, sizeof(hipfftComplex) * size);
    hipHostMalloc((void**)&h_out, sizeof(hipfftComplex) * size);

    hipMemcpy(h_ina, c_ina, sizeof(hipfftComplex) * size, hipMemcpyDeviceToHost);
    hipMemcpy(h_inb, c_inb, sizeof(hipfftComplex) * size, hipMemcpyDeviceToHost);
    hipMemcpy(h_out, c_out, sizeof(hipfftComplex) * size, hipMemcpyDeviceToHost);

    dim3 Block(32, 16);
    dim3 Grid((dimx + Block.x - 1) / Block.x, (dimy + Block.y - 1) / Block.y);
    ComplexMulti2D<<<Grid, Block>>>(c_ina, c_inb, c_out, dimx, dimy);
    hipDeviceSynchronize();

    InverseFFT2D(c_out, out, dimx, dimy);
    hipDeviceSynchronize();

    hipFree(c_ina);
    c_ina = nullptr;
    hipFree(c_inb);
    c_inb = nullptr;
    hipFree(c_out);
    c_out = nullptr;
    
    hipHostFree(h_ina);
    h_ina = nullptr;
    hipHostFree(h_inb);
    h_inb = nullptr;
    hipHostFree(h_out);
    h_out = nullptr;
}

void PadData1D(const int raw_size, const int new_size, const float *raw_signal, float *new_signal)
{
    memcpy(new_signal, raw_signal, sizeof(float) * raw_size);
}

void PadData2D(const int dimx, const int dimy, const float *raw_signal, float *new_signal)
{
    const int new_dimx = 2 * dimx - 1;
    for (int i = 0; i < dimy; ++i)
    {
        const int raw_offset = i * dimx;
        const int new_offset = i * new_dimx;
        memcpy(&new_signal[new_offset], &raw_signal[raw_offset], sizeof(float) * dimx);
    }
}

void GetSameData(const int dimx, const int dimy, const float *pad_data, float *same_data)
{
    const int new_dimx = 2 * dimx - 1;
    const int offsetx = std::ceil((dimx - 1) / 2.0);
    const int offsety = std::ceil((dimy - 1) / 2.0);

    for (int i = 0; i < dimy; ++i)
    {
        const int offset_full = (i + offsety) * new_dimx + offsetx;
        const int offset_same = i * dimx;
        memcpy(&same_data[offset_same], &pad_data[offset_full], sizeof(float) * dimx);
    }
}

void Conv1DFFT(float *ina, float *inb, float *out, const int size)
{
    hipfftComplex *c_ina, *c_inb, *c_out;
    hipMalloc((void**)&c_ina, sizeof(hipfftComplex) * size);
    hipMalloc((void**)&c_inb, sizeof(hipfftComplex) * size);
    hipMalloc((void**)&c_out, sizeof(hipfftComplex) * size);

    ForwardFFT1D(ina, c_ina, size);
    ForwardFFT1D(inb, c_inb, size);
    hipDeviceSynchronize();

    dim3 Block(128);
    dim3 Grid((size + Block.x - 1) / Block.x);
    ComplexMulti<<<Grid, Block>>>(c_ina, c_inb, c_out, size);
    hipDeviceSynchronize();

    InverseFFT1D(c_out, out, size);
    hipDeviceSynchronize();

    hipFree(c_ina);
    c_ina = nullptr;
    hipFree(c_inb);
    c_inb = nullptr;
    hipFree(c_out);
    c_out = nullptr;
}

void Conv2D(float *signal, float *kernel, const int dimx, const int dimy, float *result)
{
    const int new_dimx = 2 * dimx - 1;
    const int new_dimy = 2 * dimy - 1;
    const int new_size = new_dimx * new_dimy;

    float *h_new_signal, *h_new_kernel;
    hipHostMalloc((void**)&h_new_signal, sizeof(float) * new_size); memset(h_new_signal, 0, sizeof(float) * new_size);
    hipHostMalloc((void**)&h_new_kernel, sizeof(float) * new_size); memset(h_new_kernel, 0, sizeof(float) * new_size);

    PadData2D(dimx, dimy, signal, h_new_signal);
    PadData2D(dimx, dimy, kernel, h_new_kernel);

    float *d_signal, *d_kernel, *d_result;
    hipMalloc((void**)&d_signal, sizeof(float) * new_size); hipMemset(d_signal, 0, sizeof(float) * new_size);
    hipMalloc((void**)&d_kernel, sizeof(float) * new_size); hipMemset(d_kernel, 0, sizeof(float) * new_size);
    hipMalloc((void**)&d_result, sizeof(float) * new_size); hipMemset(d_result, 0, sizeof(float) * new_size);
    
    hipMemcpy(d_signal, h_new_signal, sizeof(float) * new_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_new_kernel, sizeof(float) * new_size, hipMemcpyHostToDevice);

    Conv2DFFT(d_signal, d_kernel, d_result, new_dimx, new_dimy);

    float *h_result;
    hipHostMalloc((void**)&h_result, sizeof(float) * new_size); 
    memset(h_result, 0, sizeof(float) * new_size);
    hipMemcpy(h_result, d_result, sizeof(float) * new_size, hipMemcpyDeviceToHost);

    GetSameData(dimx, dimy, h_result, result);

    hipFree(d_signal);
    d_signal = nullptr;
    hipFree(d_kernel);
    d_kernel = nullptr;
    hipFree(d_result);
    d_result = nullptr;

    hipHostFree(h_new_signal);
    h_new_signal = nullptr;
    hipHostFree(h_new_kernel);
    h_new_kernel = nullptr;
    hipHostFree(h_result);
    h_result = nullptr;
}

void cuConvDemo2D()
{
    const int dimx = 4;
    const int dimy = 2;
    const int size = dimx * dimy;

    float *signal, *kernel, *result;
    hipHostMalloc((void**)&signal, sizeof(float) * size); memset(signal, 0, sizeof(float) * size);
    hipHostMalloc((void**)&kernel, sizeof(float) * size); memset(kernel, 0, sizeof(float) * size);
    hipHostMalloc((void**)&result, sizeof(float) * size); memset(result, 0, sizeof(float) * size);

    for (int i = 0; i < dimx; ++i)
    {
        for (int j = 0; j < dimy; ++j)
        {
            const int index = j * dimx + i;
            signal[index] = i + 1;
            kernel[index] = i + 1;
        }
    }

    for (int i = 0; i < dimy; ++i)
    {
        const int offset = i * dimx;
        std::cout << "conv result: " << PrintArray(signal + offset, dimx) << std::endl;
    }

    // FILE *fp;
    // fp = fopen("./down_image.raw", "rb");
    // fread(signal, sizeof(float), size, fp);
    // fclose(fp);
    
    // fp = fopen("./Func0.raw", "rb");
    // fread(kernel, sizeof(float), size, fp);
    // fclose(fp);

    auto start = std::chrono::steady_clock::now();

    Conv2D(signal, kernel, dimx, dimy, result);

    auto end = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "conv fft time: " << time << " ms" << std::endl;

    for (int i = 0; i < dimy; ++i)
    {
        const int offset = i * dimx;
        std::cout << "conv result: " << PrintArray(result + offset, dimx) << std::endl;
    }

    // free memory
    hipHostFree(signal);
    signal = nullptr;
    hipHostFree(kernel);
    kernel = nullptr;
    hipHostFree(result);
    result = nullptr;
}

void cuFFTDemo1D()
{
    const int size = 4;
    const int new_size = 2 * size - 1;
    // const int kernel_size = 11;

    float *h_signal, *h_kernal, *h_result;
    hipHostMalloc((void**)&h_signal, sizeof(float) * new_size); memset(h_signal, 0, sizeof(float) * new_size);
    hipHostMalloc((void**)&h_kernal, sizeof(float) * new_size); memset(h_kernal, 0, sizeof(float) * new_size);
    hipHostMalloc((void**)&h_result, sizeof(float) * new_size); memset(h_result, 0, sizeof(float) * new_size);
    
    for (int i = 0; i < size; ++i)
    {
        h_signal[i] = 1 + i;
        h_kernal[i] = 5 + i;
    }

    std::cout << "signal initialize: " << PrintArray(h_signal, size) << std::endl;
    std::cout << "kernel initialize: " << PrintArray(h_kernal, size) << std::endl;

    // device memory
    float *d_signal, *d_kernel, *d_result;
    hipMalloc((void**)&d_signal, sizeof(float) * new_size); 
    hipMemset(d_signal, 0, sizeof(float) * new_size);
    hipMalloc((void**)&d_kernel, sizeof(float) * new_size);
    hipMemset(d_kernel, 0, sizeof(float) * new_size);
    hipMalloc((void**)&d_result, sizeof(float) * new_size);
    hipMemset(d_result, 0, sizeof(float) * new_size);
    
    hipMemcpy(d_signal, h_signal, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernal, sizeof(float) * size, hipMemcpyHostToDevice);

    // cufft
    Conv1DFFT(d_signal, d_kernel, d_result, new_size);
    hipMemcpy(h_result, d_result, sizeof(float) * new_size, hipMemcpyDeviceToHost);

    std::cout << "conv result: " << PrintArray(h_result, new_size) << std::endl;

    // free memory
    hipFree(d_signal);
    d_signal = nullptr;
    hipFree(d_kernel);
    d_kernel = nullptr;
    hipFree(d_result);
    d_result = nullptr;
    hipHostFree(h_signal);
    h_signal = nullptr;
    hipHostFree(h_kernal);
    h_kernal = nullptr;
    hipHostFree(h_result);
    h_result = nullptr;
}

int main(int argc, char **argv)
{
    // cuFFTDemo1D();

    cuConvDemo2D();

    return 0;
}