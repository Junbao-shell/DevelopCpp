#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////
/// @copyright copyright description
/// 
/// @brief cuda fft demo
/// 
/// @file cu_fft_demo.cu
/// 
/// @author GaoJunbao(junbaogao@foxmail.com)
/// 
/// @date 2022-06-14
///////////////////////////////////////////////////////////

// Current Cpp header
// System header
// C/C++ standard library header
#include <iostream>
#include <sstream>
// External library header
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
// Current module header
// Root directory header

typedef float2 Complex;

struct ComplexMultiply
{
    ComplexMultiply(int n) : N(n) {}

    __host__ __device__ Complex operator() (const Complex &a, const Complex &b)
    {
        Complex c;
        c.x = (a.x * b.x - a.y * b.y) / N;
        c.y = (a.x * b.y + a.y * b.x) / N;
        return c;
    }

    int N;
};

static __global__ void ComplexMulti(const Complex *a, const Complex *b, Complex *c, const int size)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        c[tid].x = (a[tid].x * b[tid].x - a[tid].y * b[tid].y) / size;
        c[tid].y = (a[tid].x * b[tid].y + a[tid].y * b[tid].x) / size;
    }
}

static __global__ void ComplexMulti2D(const Complex *a, const Complex *b, Complex *c, const int dimx, const int dimy)
{
    const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if (tidx < dimx && tidy < dimy)
    {
        const int i = tidy * dimx + tidx;

        c[i].x = (a[i].x * b[i].x - a[i].y * b[i].y) / (dimx * dimy);
        c[i].y = (a[i].x * b[i].y + a[i].y * b[i].x) / (dimx * dimy);
    }
}

template<typename T>
static inline std::string PrintArray(T *arr, const int size)
{
    std::stringstream stream;
    for (int i = 0; i < (size - 1); ++i)
    {
        stream << arr[i];
        stream << ' ';
    }
    stream << arr[size - 1];
    return stream.str();
}

void ForwardFFT1D(float *in, Complex *out, const int size)
{
    hipfftHandle plan;
    hipfftPlan1d(&plan, size, HIPFFT_R2C, 1);
    hipfftExecR2C(plan, in, out);
    hipfftDestroy(plan);
}

void InverseFFT1D(Complex *in, float *out, const int size)
{
    hipfftHandle plan;
    hipfftPlan1d(&plan, size, HIPFFT_C2R, 1);
    hipfftExecC2R(plan, in, out);
    hipfftDestroy(plan);
}

void ForwardFFT2D(float *in, Complex *out, const int dimx, const int dimy)
{
    hipfftHandle plan;
    hipfftPlan2d(&plan, dimy, dimx, HIPFFT_R2C);
    hipfftExecR2C(plan, in, out);
    hipfftDestroy(plan);
}

void InverseFFT2D(Complex *in, float *out, const int dimx, const int dimy)
{
    hipfftHandle plan;
    hipfftPlan2d(&plan, dimy, dimx, HIPFFT_C2R);
    hipfftExecC2R(plan, in, out);
    hipfftDestroy(plan);
}

void Conv2DFFT(float *ina, float *inb, float *out, const int dimx, const int dimy)
{
    const int size = dimx * dimy;
    Complex *c_ina, *c_inb, *c_out;
    hipMalloc((void**)&c_ina, sizeof(Complex) * size);
    hipMalloc((void**)&c_inb, sizeof(Complex) * size);
    hipMalloc((void**)&c_out, sizeof(Complex) * size);

    ForwardFFT2D(ina, c_ina, dimx, dimy);
    ForwardFFT2D(inb, c_inb, dimx, dimy);
    hipDeviceSynchronize();


    Complex *h_ina, *h_inb, *h_out;
    hipHostMalloc((void**)&h_ina, sizeof(Complex) * size);
    hipHostMalloc((void**)&h_inb, sizeof(Complex) * size);
    hipHostMalloc((void**)&h_out, sizeof(Complex) * size);

    hipMemcpy(h_ina, c_ina, sizeof(Complex) * size, hipMemcpyDeviceToHost);
    hipMemcpy(h_inb, c_inb, sizeof(Complex) * size, hipMemcpyDeviceToHost);
    hipMemcpy(h_out, c_out, sizeof(Complex) * size, hipMemcpyDeviceToHost);
    
    for (int i = 0; i < size; ++i)
    {
        std::cout << "index: " << i << " (" << h_ina[i].x << ", " << h_ina[i].y << ")" << std::endl;
    }

    dim3 Block(32, 16);
    dim3 Grid((dimx + Block.x - 1) / Block.x, (dimy + Block.y - 1) / Block.y);
    ComplexMulti2D<<<Grid, Block>>>(c_ina, c_inb, c_out, dimx, dimy);
    hipDeviceSynchronize();

    InverseFFT2D(c_out, out, dimx, dimy);
    hipDeviceSynchronize();

    hipFree(c_ina);
    c_ina = nullptr;
    hipFree(c_inb);
    c_inb = nullptr;
    hipFree(c_out);
    c_out = nullptr;
    
    hipHostFree(h_ina);
    h_ina = nullptr;
    hipHostFree(h_inb);
    h_inb = nullptr;
    hipHostFree(h_out);
    h_out = nullptr;
}

void Conv1DFFT(float *ina, float *inb, float *out, const int size)
{
    Complex *c_ina, *c_inb, *c_out;
    hipMalloc((void**)&c_ina, sizeof(Complex) * size);
    hipMalloc((void**)&c_inb, sizeof(Complex) * size);
    hipMalloc((void**)&c_out, sizeof(Complex) * size);

    ForwardFFT1D(ina, c_ina, size);
    ForwardFFT1D(inb, c_inb, size);
    hipDeviceSynchronize();

    dim3 Block(128);
    dim3 Grid((size + Block.x - 1) / Block.x);
    ComplexMulti<<<Grid, Block>>>(c_ina, c_inb, c_out, size);
    hipDeviceSynchronize();

    InverseFFT1D(c_out, out, size);
    hipDeviceSynchronize();

    hipFree(c_ina);
    c_ina = nullptr;
    hipFree(c_inb);
    c_inb = nullptr;
    hipFree(c_out);
    c_out = nullptr;
}

void cuFFTDemo2D()
{
    const int dimx = 4;
    const int dimy = 2;
    const int size = dimx * dimy;

    float *h_signal, *h_kernel, *h_result;
    hipHostMalloc((void**)&h_signal, sizeof(float) * size);
    hipHostMalloc((void**)&h_kernel, sizeof(float) * size);
    hipHostMalloc((void**)&h_result, sizeof(float) * size);
    
    for (int i = 0; i < size; ++i)
    {
        h_signal[i] = 1 + i;
        h_kernel[i] = 1 + i;
    }

    memset(h_result, 0, sizeof(float) * size);
    std::cout << "signal initialize: " << PrintArray(h_signal, size) << std::endl;
    std::cout << "kernel initialize: " << PrintArray(h_kernel, size) << std::endl;

    // device memory
    float *d_signal, *d_kernel, *d_result;
    hipMalloc((void**)&d_signal, sizeof(float) * size);
    hipMalloc((void**)&d_kernel, sizeof(float) * size);
    hipMalloc((void**)&d_result, sizeof(float) * size);
    
    hipMemcpy(d_signal, h_signal, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemset(d_result, 0, sizeof(float) * size);

    // cufft
    Conv2DFFT(d_signal, d_kernel, d_result, dimx, dimy);
    hipMemcpy(h_result, d_result, sizeof(float) * size, hipMemcpyDeviceToHost);

    std::cout << "conv result: " << PrintArray(h_result, dimx) << std::endl;
    std::cout << "conv result: " << PrintArray(h_result + dimx, dimx) << std::endl;

    // free memory
    hipFree(d_signal);
    d_signal = nullptr;
    hipFree(d_kernel);
    d_kernel = nullptr;
    hipFree(d_result);
    d_result = nullptr;
    hipHostFree(h_signal);
    h_signal = nullptr;
    hipHostFree(h_kernel);
    h_kernel = nullptr;
    hipHostFree(h_result);
    h_result = nullptr;
}

void cuFFTDemo1D()
{
    const int size = 4;
    // const int kernel_size = 11;

    float *h_signal, *h_kernel, *h_result;
    hipHostMalloc((void**)&h_signal, sizeof(float) * size);
    hipHostMalloc((void**)&h_kernel, sizeof(float) * size);
    hipHostMalloc((void**)&h_result, sizeof(float) * size);
    
    for (int i = 0; i < size; ++i)
    {
        h_signal[i] = 1 + i;
        h_kernel[i] = 5 + i;
    }

    memset(h_result, 0, sizeof(float) * size);
    std::cout << "signal initialize: " << PrintArray(h_signal, size) << std::endl;
    std::cout << "kernel initialize: " << PrintArray(h_kernel, size) << std::endl;

    // device memory
    float *d_signal, *d_kernel, *d_result;
    hipMalloc((void**)&d_signal, sizeof(float) * size);
    hipMalloc((void**)&d_kernel, sizeof(float) * size);
    hipMalloc((void**)&d_result, sizeof(float) * size);
    
    hipMemcpy(d_signal, h_signal, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemset(d_result, 0, sizeof(float) * size);

    // cufft
    Conv1DFFT(d_signal, d_kernel, d_result, size);
    hipMemcpy(h_result, d_result, sizeof(float) * size, hipMemcpyDeviceToHost);

    std::cout << "conv result: " << PrintArray(h_result, size) << std::endl;

    // free memory
    hipFree(d_signal);
    d_signal = nullptr;
    hipFree(d_kernel);
    d_kernel = nullptr;
    hipFree(d_result);
    d_result = nullptr;
    hipHostFree(h_signal);
    h_signal = nullptr;
    hipHostFree(h_kernel);
    h_kernel = nullptr;
    hipHostFree(h_result);
    h_result = nullptr;
}

int main(int argc, char **argv)
{
    // cuFFTDemo1D();

    cuFFTDemo2D();

    return 0;
}